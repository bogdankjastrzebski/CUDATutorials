// 2.cu

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void sgemm_naive(int M, int N, int K,  
							float alpha,
							const float *A, 
							const float *B,
							float beta,
							float *C) {

	const uint x = blockIdx.x * blockDim.x + threadIdx.x; 
	const uint y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < M && y < N) {
		float tmp = 0.0; 
		for (int i = 0; i < K; ++i) {
			tmp += A[x * K + i] * B[i * N + y];
		}
		C[x * N + y] = alpha * tmp + beta * C[x * N + y];
	}
}

void fill(float* M, int size, float value) {
	for (int i = 0; i < size; i++) {
		M[i] = value; 
	}
}

int CEIL_DIV(int a, int b) {
	int c = a / b; 
	int d = a % b; 
	return c + (d > 0);
}

int main() {
	const int M = 1024,
			  N = 1024,
	          K = 1024;	
	
	float *A, *d_A,
		  *B, *d_B,
          *C, *d_C;

	float alpha = 1.0f,
		  beta  = 1.0f;
				
	A = (float*)malloc(sizeof(float) * M * K);
	B = (float*)malloc(sizeof(float) * K * N);
	C = (float*)malloc(sizeof(float) * M * N);
		
	fill(A, M * K, 1.0f); 
	fill(B, K * N, 1.0f); 
	fill(C, M * N, 0.1024f); 

	hipMalloc((void**)&d_A, sizeof(float) * M * K); 
	hipMalloc((void**)&d_B, sizeof(float) * K * N);	
	hipMalloc((void**)&d_C, sizeof(float) * M * N); 

	hipMemcpy(d_A, A, sizeof(float) * M * K, hipMemcpyHostToDevice); 	
	hipMemcpy(d_B, B, sizeof(float) * K * N, hipMemcpyHostToDevice); 	
	hipMemcpy(d_C, C, sizeof(float) * M * N, hipMemcpyHostToDevice);	
		
	dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
	dim3 blockDim(32, 32, 1);	
	
	sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);	
	// cudaMemcpy(A, d_A, sizeof(float) * M * K, cudaMemcpyDeviceToHost); 	
	// cudaMemcpy(B, d_B, sizeof(float) * K * N, cudaMemcpyDeviceToHost); 	
	hipMemcpy(C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);	

	printf("C[0]: %0.4f\n", C[0]);

	printf("Done.\n");	

}




