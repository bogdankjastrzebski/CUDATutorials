// 4.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#define BN 64
#define BM 64 
#define BK 8
#define TM 8

__global__ void sgemm_naive(int M, int N, int K,  
							float alpha,
							const float *A, 
							const float *B,
							float beta,
							float *C) {
	
	// A and B are written row-wise.


	const uint innerColA = threadIdx.x % BK;
	const uint innerRowA = threadIdx.x / BK; 
	
	const uint innerColB = threadIdx.x % BN;
	const uint innerRowB = threadIdx.x / BN; 
	
	// const int numBlockSteps = (K / CHUNKSIZE) + ((K % CHUNKSIZE) > 0); 
	
	const uint cRow = blockIdx.y;
	const uint cCol = blockIdx.x; 
	
	const int threadCol = threadIdx.x % BN; 
	const int threadRow = threadIdx.x / BN;	

	const uint x = cRow * BM + innerRowA; 
	const uint y = cCol * BN + innerColB;
		

	if (x < M && y < N) {
		
		// Jump to starting position.	
		A += cRow * BM * K; // cRow * CHUNKSIZE is actual row, * K because A is row-wise; 
		B += cCol * BN; // Jump to a correct column. 
		C += cRow * BM * N + cCol * BN; // cRow * CHUNKSIZE * N moves to correct row (N is number of columns in C), cCol * CHUNKSIZE moves to correct column 
		
		__shared__ float As[BM * BK];
		__shared__ float Bs[BK * BN];	

		float threadResults[TM] = {0.0}; // Values
		// float tmp = 0.0; // the value 
		for (int bkIdx = 0; bkIdx < K; bkIdx += BK) { 
				
			// Here we want coalescing, each thread copies one value. 	
			As[innerRowA * BK + innerColA]	= A[innerRowA * K + innerColA];
			Bs[innerRowB * BN + innerColB]	= B[innerRowB * N + innerColB];
			
			// Sync after copying			
			__syncthreads(); 
			
			A += BK; // CHUNKSiZE is BK  
			B += BK * N; 	
			
			for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
				
				float Btmp = Bs[dotIdx * BN + threadCol];
				for (uint resIdx = 0; resIdx < TM; ++resIdx) {	
					threadResults[resIdx] +=
						As[(threadRow * TM + resIdx) * BK + dotIdx] * Btmp;	
				}
			}
			
			__syncthreads(); 
		}
		
		for (uint resIdx = 0; resIdx < TM; ++resIdx) {	
			C[(threadRow * TM + resIdx) * N + threadCol] =
				alpha * threadResults[resIdx] +
				beta * C[(threadRow * TM + resIdx) * N + threadCol];
		}
	}
}

void fill(float* M, int size, float value) {
	for (int i = 0; i < size; i++) {
		M[i] = value; 
	}
}

int CEIL_DIV(int a, int b) {
	int c = a / b; 
	int d = a % b; 
	return c + (d > 0);
}

bool check(float *array, int size, float value, float eps) {
	for (int idx = 0; idx < size; idx++) {
		if (abs(array[idx] - value) > eps) {
			return false;
		}
	}
	return true; 
}

float min(float *array, int size) {
	float ret = array[0]; 
	for (int i = 1; i < size; i++) {
		ret = ret < array[i] ? ret : array[i];
	}
	return ret; 
}

float max(float *array, int size) {
	float ret = array[0]; 
	for (int i = 1; i < size; i++) {
		ret = ret > array[i] ? ret : array[i];
	}
	return ret; 
}

int main() {
	const int M = 4096,
			  N = 4096,
	          K = 4096;	
	
	float *A, *d_A,
		  *B, *d_B,
          *C, *d_C;

	float alpha = 1.0f,
		  beta  = 1.0f;
				
	A = (float*)malloc(sizeof(float) * M * K);
	B = (float*)malloc(sizeof(float) * K * N);
	C = (float*)malloc(sizeof(float) * M * N);
		
	fill(A, M * K, 1.0f); 
	fill(B, K * N, 1.0f); 
	fill(C, M * N, 0.4096f); 

	hipMalloc((void**)&d_A, sizeof(float) * M * K); 
	hipMalloc((void**)&d_B, sizeof(float) * K * N);	
	hipMalloc((void**)&d_C, sizeof(float) * M * N); 

	hipMemcpy(d_A, A, sizeof(float) * M * K, hipMemcpyHostToDevice); 	
	hipMemcpy(d_B, B, sizeof(float) * K * N, hipMemcpyHostToDevice); 	
	hipMemcpy(d_C, C, sizeof(float) * M * N, hipMemcpyHostToDevice);	
		
	dim3 gridDim(CEIL_DIV(M, BM), CEIL_DIV(N, BN), 1);
	dim3 blockDim(BM * BN / TM);	
	
	sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);	
	// cudaMemcpy(A, d_A, sizeof(float) * M * K, cudaMemcpyDeviceToHost); 	
	// cudaMemcpy(B, d_B, sizeof(float) * K * N, cudaMemcpyDeviceToHost); 	
	hipMemcpy(C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);	

	printf("C[0]: %0.4f\n", C[0]);
	
	if (check(C, M * N, 4096.4096f, 0.001f)) {
		printf("Ok. \n");
	} else {
		printf("Not ok. \n");
		printf("min: %f  max: %f", min(C, M*N), max(C, M*N)); 
	} 
	
	printf("Done.\n");	

}




