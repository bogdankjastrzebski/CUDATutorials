// naive.cu / 1.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void sgemm_naive(int M, int N, int K,  
							float alpha,
							const float *A, 
							const float *B,
							float beta,
							float *C) {

	const uint x = blockIdx.x * blockDim.x + threadIdx.x; 
	const uint y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < M && y < N) {
		float tmp = 0.0; 
		for (int i = 0; i < K; ++i) {
			tmp += A[x * K + i] * B[i * N + y];
		}
		C[x * N + y] = alpha * tmp + beta * C[x * N + y];
	}
}

void fill(float* M, int size, float value) {
	for (int i = 0; i < size; i++) {
		M[i] = value; 
	}
}

int CEIL_DIV(int a, int b) {
	int c = a / b; 
	int d = a % b; 
	return c + (d > 0);
}

bool check(float *array, int size, float value, float eps) {
	for (int idx = 0; idx < size; idx++) {
		if (abs(array[idx] - value) > eps) {
			return false;
		}
	}
	return true; 
}

int main() {
	const int M = 4096,
			  N = 4096,
	          K = 4096;	
	
	float *A, *d_A,
		  *B, *d_B,
          *C, *d_C;

	float alpha = 1.0f,
		  beta  = 1.0f;
				
	A = (float*)malloc(sizeof(float) * M * K);
	B = (float*)malloc(sizeof(float) * K * N);
	C = (float*)malloc(sizeof(float) * M * N);
		
	fill(A, M * K, 1.0f); 
	fill(B, K * N, 1.0f); 
	fill(C, M * N, 0.4096f); 

	hipMalloc((void**)&d_A, sizeof(float) * M * K); 
	hipMalloc((void**)&d_B, sizeof(float) * K * N);	
	hipMalloc((void**)&d_C, sizeof(float) * M * N); 

	hipMemcpy(d_A, A, sizeof(float) * M * K, hipMemcpyHostToDevice); 	
	hipMemcpy(d_B, B, sizeof(float) * K * N, hipMemcpyHostToDevice); 	
	hipMemcpy(d_C, C, sizeof(float) * M * N, hipMemcpyHostToDevice);	
		
	dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
	dim3 blockDim(32, 32, 1);	
	
	sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);	
	// cudaMemcpy(A, d_A, sizeof(float) * M * K, cudaMemcpyDeviceToHost); 	
	// cudaMemcpy(B, d_B, sizeof(float) * K * N, cudaMemcpyDeviceToHost); 	
	hipMemcpy(C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);	

	printf("C[0]: %0.4f\n", C[0]);
	
	if (check(C, M * N, 4096.4096f, 0.0001f)) {
		printf("Ok. \n");
	} else {
		printf("Not ok. \n");
	} 

	printf("Done.\n");	

}




