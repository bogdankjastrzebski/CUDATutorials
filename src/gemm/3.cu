// 2.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#define BLOCKSIZE 32

__global__ void sgemm_naive(int M, int N, int K,  
							float alpha,
							const float *A, 
							const float *B,
							float beta,
							float *C) {

	const uint x = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE); 
	const uint y = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);
	
	if (x < M && y < N) {
		float tmp = 0.0; 
		for (int i = 0; i < K; ++i) {
			tmp += A[x * K + i] * B[i * N + y];
		}
		C[x * N + y] = alpha * tmp + beta * C[x * N + y];
	}
}

void fill(float* M, int size, float value) {
	for (int i = 0; i < size; i++) {
		M[i] = value; 
	}
}

int CEIL_DIV(int a, int b) {
	int c = a / b; 
	int d = a % b; 
	return c + (d > 0);
}

bool check(float *array, int size, float value, float eps) {
	for (int idx = 0; idx < size; idx++) {
		if (abs(array[idx] - value) > eps) {
			return false;
		}
	}
	return true; 
}

float min(float *array, int size) {
	float ret = array[0]; 
	for (int i = 1; i < size; i++) {
		ret = ret < array[i] ? ret : array[i];
	}
	return ret; 
}

float max(float *array, int size) {
	float ret = array[0]; 
	for (int i = 1; i < size; i++) {
		ret = ret > array[i] ? ret : array[i];
	}
	return ret; 
}

int main() {
	const int M = 4096,
			  N = 4096,
	          K = 4096;	
	
	float *A, *d_A,
		  *B, *d_B,
          *C, *d_C;

	float alpha = 1.0f,
		  beta  = 1.0f;
				
	A = (float*)malloc(sizeof(float) * M * K);
	B = (float*)malloc(sizeof(float) * K * N);
	C = (float*)malloc(sizeof(float) * M * N);
		
	fill(A, M * K, 1.0f); 
	fill(B, K * N, 1.0f); 
	fill(C, M * N, 0.4096f); 

	hipMalloc((void**)&d_A, sizeof(float) * M * K); 
	hipMalloc((void**)&d_B, sizeof(float) * K * N);	
	hipMalloc((void**)&d_C, sizeof(float) * M * N); 

	hipMemcpy(d_A, A, sizeof(float) * M * K, hipMemcpyHostToDevice); 	
	hipMemcpy(d_B, B, sizeof(float) * K * N, hipMemcpyHostToDevice); 	
	hipMemcpy(d_C, C, sizeof(float) * M * N, hipMemcpyHostToDevice);	
		
	dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
	dim3 blockDim(32*32);	
	
	sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);	
	// cudaMemcpy(A, d_A, sizeof(float) * M * K, cudaMemcpyDeviceToHost); 	
	// cudaMemcpy(B, d_B, sizeof(float) * K * N, cudaMemcpyDeviceToHost); 	
	hipMemcpy(C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);	

	printf("C[0]: %0.4f\n", C[0]);
	
	if (check(C, M * N, 4096.4096f, 0.001f)) {
		printf("Ok. \n");
	} else {
		printf("Not ok. \n");
		printf("min: %f  max: %f", min(C, M*N), max(C, M*N)); 
	} 
	
	printf("Done.\n");	

}




